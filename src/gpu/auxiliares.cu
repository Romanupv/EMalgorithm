
#include <hip/hip_runtime.h>
#include <limits>

using namespace std;

// Constantes
const double MENOS_INFINITO = -numeric_limits<double>::max();
const size_t BLOCK_SIZE = 128;

__device__ unsigned int contadorBloques = 0;

__device__ double logaritmoDeterminante(double *g_L, const size_t k, const size_t numDimensiones)
{
    double suma = 0.0;

    for (size_t j = 0; j < numDimensiones; j++) {
        suma += log(g_L[k * numDimensiones * numDimensiones + j * numDimensiones + j]);
    }

    return 2.0 * suma;
}

template <size_t blockSize>
__device__ void reducirBloque(volatile double *sharedData, double suma, const size_t tid)
{
    sharedData[tid] = suma;

    __syncthreads();

    if (blockSize >= 512) {
        if (tid < 256) {
            sharedData[tid] = suma = suma + sharedData[tid + 256];
        }

        __syncthreads();
    }

    if (blockSize >= 256) {
        if (tid < 128) {
            sharedData[tid] = suma = suma + sharedData[tid + 128];
        }

        __syncthreads();
    }

    if (blockSize >= 128) {
        if (tid < 64) {
            sharedData[tid] = suma = suma + sharedData[tid + 64];
        }

        __syncthreads();
    }

    if (tid < 32) {
        if (blockSize >= 64) {
            sharedData[tid] = suma = suma + sharedData[tid + 32];
        }

        if (blockSize >= 32) {
            sharedData[tid] = suma = suma + sharedData[tid + 16];
        }

        if (blockSize >= 16) {
            sharedData[tid] = suma = suma + sharedData[tid + 8];
        }

        if (blockSize >= 8) {
            sharedData[tid] = suma = suma + sharedData[tid + 4];
        }

        if (blockSize >= 4) {
            sharedData[tid] = suma = suma + sharedData[tid + 2];
        }

        if (blockSize >= 2) {
            sharedData[tid] = suma = suma + sharedData[tid + 1];
        }
    }
}

template <size_t blockSize, typename Predicate, typename Predicate2>
__device__ void reducirFinal(Predicate valor, Predicate2 direccionResultado, volatile double *sharedData, size_t numTrozos)
{
    const size_t tid = threadIdx.x;
    double suma = 0.0;
    int i = tid;

    while (i < numTrozos)
    {
        suma += *(valor(i));
        i += blockSize;
    }

    reducirBloque<blockSize>(sharedData, suma, tid);

    if (tid == 0) {
        *(direccionResultado()) = sharedData[0];
    }
}

template <size_t blockSize, typename Predicate, typename Predicate2, typename Predicate3>
__device__ void reducir(Predicate valor, Predicate2 direccionResultado, Predicate3 reduccionFinal, const size_t n, volatile double *sharedData, const size_t numBloques)
{
    __shared__ bool esUltimoBloque;

    const size_t tid = threadIdx.x;
    const size_t gridSize = (blockSize * 2) * gridDim.x;

    size_t i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    double suma = 0.0;

    while (i < n) {
        suma += valor(i);

        if (i + blockSize < n) {
            suma += valor(i+blockSize);
        }

        i += gridSize;
    }

    reducirBloque<blockSize>(sharedData, suma, tid);

    if (tid == 0) {
    	*(direccionResultado()) = sharedData[0];

        __threadfence();

        unsigned int ticket = atomicInc(&contadorBloques, numBloques);
        esUltimoBloque = (ticket == numBloques - 1);
    }

    __syncthreads();

    if (esUltimoBloque) {
        reduccionFinal();

        if (tid == 0) {
            contadorBloques = 0;
        }
    }
}
